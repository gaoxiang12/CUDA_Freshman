
#include <hip/hip_runtime.h>
#include <stdio.h>

/// __global__ device 上的函数
__global__ void hello_world(void) { printf("GPU: Hello world!\n"); }

int main(int argc, char **argv) {
    printf("CPU: Hello world!\n");
    hello_world<<<1, 10>>>();  // 10个线程

    /// 与GPU同步，防止主线程退出
    // if no this line ,it can not output hello world from gpu
    hipDeviceReset();
    return 0;
}
