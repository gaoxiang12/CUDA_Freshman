#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__ void checkIndex(void) {
    printf(
        "threadIdx:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d)\
  gridDim(%d,%d,%d)\n",
        threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
        gridDim.x, gridDim.y, gridDim.z);
}
int main(int argc, char **argv) {
    /// 从大到小： grid, block, thread

    int nElem = 6;
    dim3 block(3);
    dim3 grid((nElem + block.x - 1) / block.x);
    printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);

    std::cout << std::endl;
    checkIndex<<<grid, block>>>();
    hipDeviceReset();
    return 0;
}
